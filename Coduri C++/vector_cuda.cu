#include <iostream>
#include <hip/hip_runtime.h>

__global__ void increment_vector(int* vec, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        vec[idx] += 1;
    }
}

int main() {
    const int N = 10000000; // 100 milioane
    size_t size = N * sizeof(int);

    int* h_vec = (int*)malloc(size);
    int* d_vec;

    // Inițializăm vectorul pe host
    for (int i = 0; i < N; i++) h_vec[i] = 0;

    hipMalloc(&d_vec, size);
    hipMemcpy(d_vec, h_vec, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Rulăm kernelul CUDA
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    increment_vector<<<blocksPerGrid, threadsPerBlock>>>(d_vec, N);

    hipMemcpy(h_vec, d_vec, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Timp de executie (CUDA): " << milliseconds / 1000.0f << " secunde\n";

    hipFree(d_vec);
    free(h_vec);

    return 0;
}
